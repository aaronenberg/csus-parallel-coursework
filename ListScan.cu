#include "hip/hip_runtime.h"
#include <wb.h>

#define BLOCK_SIZE 512 

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void
scan(float *input, float *output, float *aux, int len)
{
    //@@ Modify the body of this kernel to generate the scanned blocks
    //@@ Make sure to use the workefficient version of the parallel scan
    //@@ Also make sure to store the block sum to the aux array 
    const int SECTION_SIZE = BLOCK_SIZE << 1;
    __shared__ float IO[SECTION_SIZE]; 

    int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len)
        IO[threadIdx.x] = input[i];
    else
        IO[threadIdx.x] = 0.0f;
    if (i + blockDim.x < len)
        IO[threadIdx.x + blockDim.x] = input[i + blockDim.x];
    else
        IO[threadIdx.x + blockDim.x] = 0.0f;

    for (unsigned int stride = 1; stride <= BLOCK_SIZE; stride *= 2)
    {
        __syncthreads();
        int idx = (threadIdx.x + 1) * 2 * stride - 1;
        if (idx < SECTION_SIZE) 
            IO[idx] += IO[idx - stride];
    }

    for (int stride = BLOCK_SIZE >> 1; stride > 0; stride /= 2)
    {
        __syncthreads();
        int idx = (threadIdx.x + 1) * 2 * stride - 1;
        if (idx + stride < SECTION_SIZE)
            IO[idx + stride] += IO[idx];
    }

    __syncthreads();

    if (i < len)
        output[i] = IO[threadIdx.x];
    if (i + blockDim.x < len)
        output[i + blockDim.x] = IO[threadIdx.x + blockDim.x];

    if (threadIdx.x == 0 && aux)
        aux[blockIdx.x] = IO[SECTION_SIZE - 1];
}

__global__ void
addScannedBlockSums(float *input, float *aux, int len)
{
	//@@ Modify the body of this kernel to add scanned block sums to 
	//@@ all values of the scanned blocks
    int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    if (blockIdx.x)
    {
        if (i < len)
            input[i] += aux[blockIdx.x - 1];
        if (i + blockDim.x < len)
            input[i + blockDim.x] += aux[blockIdx.x - 1];
    }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output 1D list
  float *deviceInput;
  float *deviceOutput;
  float *deviceAuxArray, *deviceAuxScannedArray;
  int numElements; // number of elements in the input/output list

  args = wbArg_read(argc, argv);
 
  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

  wbTime_start(GPU, "Allocating device memory.");
  //@@ Allocate device memory
  //you can assume that aux array size would not need to be more than BLOCK_SIZE*2 (i.e., 1024)
  int auxSize = BLOCK_SIZE * 2 * sizeof(float);
  int ioSize = numElements * sizeof(float);

  hipMalloc((void **) &deviceInput, ioSize);
  hipMalloc((void **) &deviceOutput, ioSize);
  hipMalloc((void **) &deviceAuxArray, auxSize);
  hipMalloc((void **) &deviceAuxScannedArray, auxSize);
  wbTime_stop(GPU, "Allocating device memory.");

  wbTime_start(GPU, "Clearing output device memory.");
  //@@ zero out the deviceOutput using hipMemset() by uncommenting the below line
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbTime_stop(GPU, "Clearing output device memory.");

  wbTime_start(GPU, "Copying input host memory to device.");
  //@@ Copy input host memory to device	
  hipMemcpy(deviceInput, hostInput, ioSize, hipMemcpyHostToDevice);
  hipMemcpy(deviceOutput, hostOutput, ioSize, hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input host memory to device.");

  //@@ Initialize the grid and block dimensions here
  int numBlocks = ceil((float) numElements / (BLOCK_SIZE << 1));
  dim3 dimBlock(BLOCK_SIZE, 1, 1);
  dim3 dimGrid(numBlocks, 1, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  //@@ You need to launch scan kernel twice: 1) for generating scanned blocks 
  //@@ (hint: pass deviceAuxArray to the aux parameter)
  //@@ and 2) for generating scanned aux array that has the scanned block sums. 
  //@@ (hint: pass NULL to the aux parameter)
  //@@ Then you should call addScannedBlockSums kernel.
  scan<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, deviceAuxArray, numElements);
  hipDeviceSynchronize();
  scan<<<dimGrid, dimBlock>>>(deviceAuxArray, deviceAuxScannedArray, NULL, numElements);
  hipDeviceSynchronize();
  addScannedBlockSums<<<dimGrid, dimBlock>>>(deviceOutput, deviceAuxScannedArray, numElements);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output device memory to host");
  //@@ Copy results from device to host	
  hipMemcpy(hostInput, deviceInput, ioSize, hipMemcpyDeviceToHost);
  hipMemcpy(hostOutput, deviceOutput, ioSize, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output device memory to host");

  wbTime_start(GPU, "Freeing device memory");
  //@@ Deallocate device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);
  hipFree(deviceAuxArray);
  hipFree(deviceAuxScannedArray);
  wbTime_stop(GPU, "Freeing device memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}
